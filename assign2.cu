//CSCI415 - Assignment 2
//Original by: Saeed Salem, 2/25/2015
//Updated by: Otto Borchert, 2/20/2017
//To compile: make clean; make
//To run: ./assign2

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>

typedef std::vector< std::vector<int> > AdjacencyMatrix;
AdjacencyMatrix adjMatrix;

/* void averageForNodes(int *forSum,int *miOutput, int n, float *output)
{
	float sum =0;
	float average=0;
	for(int a =0; a<n;a++)
	{
		if(forSum[a]<2){
		 sum+=0;
		}
		else
		{
		 int top = 2*miOutput[a];
		 int botton = (forSum[a]-1)*forSum[a];
		 sum += (float)top/botton; 
		}

	}
	 	average = (float)sum/totalNodes;
		output = average;


}*/

/* void countForMi(int *initialArray, int *friends, int *outputMi, int *countForRound)
{
	int countOfMi =0;
	for(int a=0; a<friends.size()-1;a++)
	{
		for(int b=1; b<friends.size();b++)
		if(initialArray[]==1)
		countOfMi++;
		}
	}
	outputMi=countOfMi;
	
}*/

void printAdjMatrix(AdjacencyMatrix adjMatrix)
{
    for (int i=0; i<adjMatrix.size(); i++)
    {
        for (int j=0; j<adjMatrix[i].size(); j++) 
        {
            std::cout << adjMatrix[i][j] << " ";
        }
        std::cout << std::endl;
    }
}

void matrixToArray(AdjacencyMatrix adjMatrix, int *initialArray)
{
	int countOfNodes = 0;
	for(int i=0; i<adjMatrix.size();i++)
	{
		for(int j=0;j<adjMatrix[i].size();j++)
		{
		initialArray[countOfNodes]=adjMatrix[i][j];
			countOfNodes++;
		}
	}
}

void clustering_serial(int *input,float *output,int n)
{
		int sum =0;
		int countForMi =0;
		int friends [n];
		int countForRound=1;
		int forSum[n];
		int miOutput[n];
	
	for(int vi=0; vi<input.size();vi++)
	{
		 sum += input[vi];
		 if(input[vi]==1)
		{
		 friends[countForMi]= vi;
		 countForMi++;
		}
		int a = vi+1;
		int b = countForRound*n;
		if(a%b==0)
		{
		 int outputMi=0;
		 forSum[countForRound-1]=sum;
		 countForMi(input,friends,outputMi,countForRound);
		 miOutput[countForRound-1]=outputMi;
		 countForRound++;
		 
		}

	}
	averageForNodes(forSum,miOutput,n,output);

}

int main()
{
    std::fstream myfile("toyGraph1.txt",std::ios_base::in);
    int u,v;
    int maxNode = 0;
    std::vector< std::pair<int,int> > allEdges;
    while(myfile >> u >> v)
    {
        allEdges.push_back(std::make_pair(u,v));
        if(u > maxNode)
          maxNode = u;

        if(v > maxNode)
          maxNode = v;                 
    }

    int n = maxNode + 1;  //Since nodes starts with 0
    std::cout << "Graph has " << n << " nodes" << std::endl;

    adjMatrix = AdjacencyMatri x(n,std::vector<int>(n,0));
    //populate the matrix
    for(int i =0; i<allEdges.size() ; i++){
       u = allEdges[i].first;
       v = allEdges[i].second;
       adjMatrix[u][v] = 1;
       adjMatrix[v][u] = 1;
    } 
    //You can also make a list of neighbors for each node if you want.
    int initialArray[n*n];
    printAdjMatrix(adjMatrix);

    //convert adjacency matrix to 1 d array
	matrixToArray(adjMatrix, initialArray);	
	
    //TODO: Write serial clustering coefficent code; include timing and error checking
	float h_cpu_result=0;
	
	clustering_serial(initialArray,&h_cpu_result,n);

    //TODO: Write parallel clustering cofficient code; include timing and error checking

    //TODO: Compare serial and parallel results

    return 0;
}
